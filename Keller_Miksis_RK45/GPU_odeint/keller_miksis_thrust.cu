#include "hip/hip_runtime.h"
//============================================================================
// Name        : keller_miksis_thrust.cu
// Author      : Lambert Plavecz
// Version     :
// Copyright   : no
// Description : Parameter study of the Keller-Miksis equation with odeint RKCK54 and Thrust
//============================================================================


#include <iostream>
#include <fstream>
#include <iomanip>
#include <string>
#include <chrono>
#define _USE_MATH_DEFINES
#include <cmath>
#include <utility>

#include <thrust/device_vector.h>
#include <thrust/reduce.h>
#include <thrust/functional.h>

#include <boost/numeric/odeint.hpp>
#include <boost/numeric/odeint/external/thrust/thrust.hpp>


using namespace std;
using namespace boost::numeric::odeint;

const double ro_L = 9.970639504998557e+02;
const double P_inf = 1.0e+5;
const double p_v = 3.166775638952003e+03;
const double sigma = 0.071977583160056;
const double R_E = 10.0/1.0e6;
const double gam = 1.4;
const double c_L = 1.497251785455527e+03;
const double mu_L = 8.902125058209557e-04;
const double theta = 0.0;

typedef double value_type;
typedef thrust::device_vector< value_type > state_type;

int num = 12; 

string file_name = "kellermiksis_thrust_output.txt";

struct keller_miksis
{
	size_t m_N;
    state_type &m_f;
	state_type C;
	
	struct param_fill_functor
    {
		
        template< class T >
        __host__ __device__
        void operator()( T t )const
        {	
            value_type omega = thrust::get< 1 >( t );
			double twr = 2*M_PI/(R_E*omega);
			double P_A1 = 1.5e5;
			double omega2 = 0.0;
			double P_A2 = 0.0;
            thrust::get< 0 >( thrust::get< 0 >( t )) = (P_inf - p_v + 2*sigma/R_E)/ro_L*twr*twr;
			thrust::get< 1 >( thrust::get< 0 >( t )) = (1-3*gam)/(ro_L*c_L)*(P_inf - p_v + 2*sigma/R_E)*twr; 
			thrust::get< 2 >( thrust::get< 0 >( t )) = (P_inf - p_v)/ro_L * twr*twr;
			thrust::get< 3 >( thrust::get< 0 >( t )) = 2*sigma/(ro_L*R_E) *twr*twr;
			thrust::get< 4 >( thrust::get< 0 >( t )) = 4*mu_L/(ro_L*R_E*R_E) * 2*M_PI/omega;
			thrust::get< 5 >( thrust::get< 0 >( t )) = P_A1/ro_L * twr*twr;
			thrust::get< 6 >( thrust::get< 0 >( t )) = P_A2/ro_L *twr*twr;
			thrust::get< 0 >( thrust::get< 2 >( t )) = R_E * omega*P_A1/(ro_L*c_L) * twr*twr;
			thrust::get< 1 >( thrust::get< 2 >( t )) = R_E * omega*P_A2/(ro_L*c_L) * twr*twr;
			thrust::get< 2 >( thrust::get< 2 >( t )) = R_E*omega/(2*M_PI*c_L);
			thrust::get< 3 >( thrust::get< 2 >( t )) = 3*gam;
			thrust::get< 4 >( thrust::get< 2 >( t )) = omega2/omega;
			thrust::get< 5 >( thrust::get< 2 >( t )) =  theta;
        }
    };

    keller_miksis( size_t N , state_type &f): m_N( N ) , m_f( f )
	{ 
		C = state_type(m_N*13); //create param vector
		//fill param vector
		thrust::for_each(
                thrust::make_zip_iterator( thrust::make_tuple(
                        thrust::make_zip_iterator( thrust::make_tuple(
							C.begin(),
							C.begin() + m_N ,
							C.begin() + 2 * m_N,
							C.begin() + 3 * m_N ,
							C.begin() + 4 * m_N , 
							C.begin() + 5 * m_N , 
							C.begin() + 6 * m_N 
						)),
                        m_f.begin() ,
                        thrust::make_zip_iterator( thrust::make_tuple(
							C.begin() + 7 * m_N,
							C.begin() + 8 * m_N ,
							C.begin() + 9 * m_N , 
							C.begin() + 10 * m_N , 
							C.begin() + 11 * m_N ,
							C.begin() + 12 * m_N 
						))
				) ) ,
                thrust::make_zip_iterator( thrust::make_tuple(
                        thrust::make_zip_iterator( thrust::make_tuple(
							C.begin() + m_N ,
							C.begin() + 2 * m_N,
							C.begin() + 3 * m_N ,
							C.begin() + 4 * m_N , 
							C.begin() + 5 * m_N , 
							C.begin() + 6 * m_N ,
							C.begin() + 7 * m_N 
						)),
                        m_f.end() ,
                        thrust::make_zip_iterator( thrust::make_tuple(
							C.begin() + 8 * m_N ,
							C.begin() + 9 * m_N , 
							C.begin() + 10 * m_N , 
							C.begin() + 11 * m_N ,
							C.begin() + 12 * m_N ,
							C.end()
						))
				) ) ,
                param_fill_functor() );
		//f is no longer needed, free it
		m_f.clear();
		m_f.shrink_to_fit();
	}
	
	struct impact_functor
    {
		
		double m_time;
		impact_functor(double time): m_time(time){}
		
        template< class T >
        __host__ __device__
        void operator()( T t )const
        {
            value_type q = thrust::get< 4 >( t );
            value_type x = thrust::get< 0 >( t );
            value_type y = thrust::get< 1 >( t );
			value_type C0 = thrust::get< 0 >( thrust::get< 5 >( t ));
			value_type C1 = thrust::get< 1 >( thrust::get< 5 >( t ));
			value_type C2 = thrust::get< 2 >( thrust::get< 5 >( t ));
			value_type C3 = thrust::get< 3 >( thrust::get< 5 >( t ));
			value_type C4 = thrust::get< 4 >( thrust::get< 5 >( t ));
			value_type C5 = thrust::get< 5 >( thrust::get< 5 >( t ));
			value_type C6 = thrust::get< 6 >( thrust::get< 5 >( t )); 
			value_type C7 = thrust::get< 0 >( thrust::get< 6 >( t ));
			value_type C8 = thrust::get< 1 >( thrust::get< 6 >( t ));
			value_type C9 = thrust::get< 2 >( thrust::get< 6 >( t ));
			value_type C10 = thrust::get< 3 >( thrust::get< 6 >( t ));
			value_type C11 = thrust::get< 4 >( thrust::get< 6 >( t ));
			value_type C12 = thrust::get< 5 >( thrust::get< 6 >( t ));
			
			double rx0 = 1.0/x;
			double N = (C0+C1*y)*pow(rx0,C10) - C2*(1.0+C9*y) -C3*rx0 -C4*y*rx0 - (1.0 - C9*y/3.0)*1.5*y*y
						-(C5*sin(2.0*M_PI*m_time) + C6*sin(2.0*M_PI*C11*m_time + C12)) * (1.0+C9*y)
						-x*(C7*cos(2.0*M_PI*m_time) + C8*cos(2.0*M_PI*C11*m_time+C12));

			double D = x - C9*x*y + C4*C9;
			
            thrust::get< 2 >( t ) = y;
            thrust::get< 3 >( t ) = N/D;
        }
    };

    template< class State , class Deriv >
    void operator()(  const State &x , Deriv &dxdt , value_type t ) const
    {
        thrust::for_each(
                thrust::make_zip_iterator( thrust::make_tuple(
                        boost::begin( x ) ,
                        boost::begin( x ) + m_N ,
						boost::begin( dxdt ) ,
                        boost::begin( dxdt ) + m_N ,
                        m_f.begin() ,
						thrust::make_zip_iterator( thrust::make_tuple(
							C.begin(),
							C.begin() + m_N ,
							C.begin() + 2 * m_N,
							C.begin() + 3 * m_N ,
							C.begin() + 4 * m_N , 
							C.begin() + 5 * m_N , 
							C.begin() + 6 * m_N 
						) ),
						thrust::make_zip_iterator( thrust::make_tuple(
							C.begin() + 7 * m_N,
							C.begin() + 8 * m_N ,
							C.begin() + 9 * m_N , 
							C.begin() + 10 * m_N , 
							C.begin() + 11 * m_N ,
							C.begin() + 12 * m_N 
						) )
				) ),
                thrust::make_zip_iterator( thrust::make_tuple(
                        boost::begin( x ) + m_N ,
                        boost::begin( x ) + 2 * m_N ,
						boost::begin( dxdt ) + m_N ,
                        boost::begin( dxdt ) + 2 * m_N ,
                        m_f.end() ,
						thrust::make_zip_iterator( thrust::make_tuple(
							C.begin() + m_N ,
							C.begin() + 2 * m_N,
							C.begin() + 3 * m_N ,
							C.begin() + 4 * m_N , 
							C.begin() + 5 * m_N , 
							C.begin() + 6 * m_N ,
							C.begin() + 7 * m_N 
						) ),
						thrust::make_zip_iterator( thrust::make_tuple(
							C.begin() + 8 * m_N ,
							C.begin() + 9 * m_N , 
							C.begin() + 10 * m_N , 
							C.begin() + 11 * m_N ,
							C.begin() + 12 * m_N ,
							C.end()
						) )
                ) ) ,
                impact_functor(t) );
    }
};

class observer
{

public:

	struct observer_functor
    {
		
		template< class T >
        __host__ __device__
        void operator()( T t )
        {
            value_type x = thrust::get< 0 >( t );
			value_type max = thrust::get< 1 >( t );
			if(x > max) thrust::get< 1 >( t ) = x; //max
        }
    };
	
    observer(size_t N, state_type &max): m_N( N ), m_max(max){}

	template< class State >
    void operator()( State &x, double t )
	//(const state_type &x , value_type t )
	{
		thrust::for_each(
                thrust::make_zip_iterator( thrust::make_tuple(
                        x.begin() ,
						m_max.begin() ) ),
                thrust::make_zip_iterator( thrust::make_tuple(
                        x.end(),
						m_max.end() ) ) ,
                obs_fun );
    }
	
private:
	state_type &m_max;
	size_t m_N;
	observer_functor obs_fun;
};
int nums[12] = {16, 32, 64, 128, 256, 512, 1024, 1536, 3072, 3840, 5120, 7680};// 15360, 30720, 46080, 61440, 76800, 92160, 122880, 184320, 307200, 768000, 4147200};

int main() {
	cout << "Keller-Miksis Thrust started" << endl;

	typedef runge_kutta_cash_karp54< state_type , value_type , state_type , value_type > stepper_type;

	for(int jj=0; jj < 12;jj++){ //parameter loop
	
	num = nums[jj];
	cout << num << endl;
	auto t1 = chrono::high_resolution_clock::now();

	thrust::host_vector< value_type > f_host(num);
	double B = 20.0;
	double E = 1000.0;
	double EpB = E/B;
	double invnum = 1.0/(num-1);
	for( size_t i=0 ; i<num ; i++){
		f_host[i] = B*pow(EpB, i*invnum)*2000.0*M_PI;
	}

	state_type f = f_host;

	state_type x( 2 * num );

	// initialize x
	thrust::fill( x.begin() , x.begin() + num , 1.0 );
	// initialize y
	thrust::fill( x.begin() + num, x.end() , 0.0 );
	
	state_type max(num);
	thrust::fill( max.begin(), max.end(), 0.0);     //arbitrary small number, initial max

	keller_miksis km( num , f );
	observer obs(num, max);
	
	auto stepper = make_controlled(1.0e-10, 1.0e-10, stepper_type());
	
	integrate_adaptive(boost::ref(stepper), boost::ref(km), x, 0.0, 1024.0, 0.01);
	
	integrate_adaptive(boost::ref(stepper), boost::ref(km), x, 1024.0, 1088.0, 0.01, obs);
	
	thrust::host_vector<value_type> max_host(num);
	max_host = max;
	
	ofstream ofs(file_name);
	if(!ofs.is_open())exit(-1);
	ofs << setprecision(17);
	for(int u = 0;u < num;u++){
		ofs << 1.5 << " " << f_host[u]/(2000.0*M_PI) << " " << 0.0 << " " << 0.0 << " " << theta << " " << R_E 
			<< " " << max_host[u] << "\n";
	}

	ofs.flush();
	ofs.close();

	auto t2 = chrono::high_resolution_clock::now();
	cout << "Done" << endl;
	cout << "Time (ms):" << std::chrono::duration_cast<std::chrono::milliseconds>(t2 - t1).count() << endl;
	
	} //end of parameter loop
	return 0;
}
