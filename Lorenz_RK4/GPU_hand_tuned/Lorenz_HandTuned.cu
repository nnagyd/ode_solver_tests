
#include <hip/hip_runtime.h>
#include <iostream>
#include <iomanip>
#include <vector>
#include <string>
#include <fstream>

using namespace std;

void Linspace(double*, double, double, int);
void Uniform(double*, double, int);

__global__ void RungeKuttaStepOriginal(double* __restrict__, const double* __restrict__, int);
__global__ void RungeKuttaStepRegisterFriendly(double* __restrict__, const double* __restrict__, int);
__device__ void Lorenz(double* __restrict__, const double* __restrict__, double);

template <class DataType>
DataType* AllocateHostMemory(int);
template <class DataType>
DataType* AllocateDeviceMemory(int);

int main()
{
// INITIAL SETUP ----------------------------------------------------------------------------------
	
	int NumberOfProblems = 768000;
	int NumberOfThreads  = NumberOfProblems;
	int BlockSize        = 64;
	
	hipSetDevice(1);
	
	double* h_State      = AllocateHostMemory<double>( 3*NumberOfProblems );
	double* h_Parameters = AllocateHostMemory<double>(   NumberOfProblems );
	double* d_State      = AllocateDeviceMemory<double>( 3*NumberOfProblems );
	double* d_Parameters = AllocateDeviceMemory<double>(   NumberOfProblems );
	
	Linspace(h_Parameters, 0.0, 21.0, NumberOfProblems);
	Uniform(h_State, 10.0, NumberOfProblems);
	Uniform(&h_State[   NumberOfProblems ], 10.0, NumberOfProblems);
	Uniform(&h_State[ 2*NumberOfProblems ], 10.0, NumberOfProblems);
	
	
	hipMemcpy(d_State, h_State, 3*sizeof(double)*NumberOfProblems, hipMemcpyHostToDevice);
	hipMemcpy(d_Parameters, h_Parameters, sizeof(double)*NumberOfProblems, hipMemcpyHostToDevice);
	
	int GridSize = NumberOfThreads/BlockSize + (NumberOfThreads % BlockSize == 0 ? 0:1);
	
	clock_t SimulationStart;
	clock_t SimulationEnd;
	
	SimulationStart = clock();
	RungeKuttaStepRegisterFriendly<<<GridSize, BlockSize>>> (d_State, d_Parameters, NumberOfProblems);
	hipDeviceSynchronize();
	SimulationEnd = clock();
	
	cout << "Simulation time: " << 1000.0*(SimulationEnd-SimulationStart) / CLOCKS_PER_SEC << "ms" << endl << endl;
	cout << "Simulation time / 1000 RK4 step: " << 1000.0*(SimulationEnd-SimulationStart) / CLOCKS_PER_SEC << "ms" << endl;
	cout << "Ensemble size:                   " << NumberOfProblems << endl << endl;
		
	hipMemcpyAsync(h_State, d_State, 3*sizeof(double)*NumberOfProblems, hipMemcpyDeviceToHost);
	
	
	//for (int i=0; i<NumberOfProblems; i++)
	//	cout << "P: " << h_Parameters[i] << "   Sates: " << h_State[i] << ", " << h_State[i+NumberOfProblems] << ", " << h_State[i+2*NumberOfProblems] << endl;
}

// AUXILIARY FUNCTION -----------------------------------------------------------------------------

void Linspace(double* x, double B, double E, int N)
{
    double Increment;
	
	x[0]   = B;
	
	if ( N>1 )
	{
		x[N-1] = E;
		Increment = (E-B)/(N-1);
		
		for (int i=1; i<N-1; i++)
		{
			x[i] = B + i*Increment;
		}
	}
}

void Uniform(double* x, double V, int N)
{
	for (int i=0; i<N; i++)
	{
		x[i] = V;
	}
}

__forceinline__ __device__ void Lorenz(double* __restrict__ F, const double* __restrict__ X, double P)
{
	// How 5 FMA and 1 ADD/MUL is possible
	F[0] = 10.0*(X[1] - X[0]); // 2 FP inst: 1 FMA, 1 ADD
	F[1] = P*X[0] - X[1] - X[0]*X[2]; // 2 FP inst: 2 FMA
	F[2] = X[0]*X[1] - 2.666 * X[2]; // 2 FP inst: 1 MUL, 1 FMA
}

__global__ void RungeKuttaStepRegisterFriendly(double* __restrict__ d_State, const double* __restrict__ d_Parameters, int N)
{
	int tid = threadIdx.x + blockIdx.x*blockDim.x;
	
	if (tid < N)
	{
		double X[3];
		double P;
		
		double k1[3];
		double ks[3];
		double x[3];
		
		double T    = 0.0;
		double dT   = 1e-3;
		double dTp2 = 0.5*dT;
		double dTp6 = dT * (1.0/6.0);
		//double t;
		
		X[0] = d_State[tid];
		X[1] = d_State[tid + N];
		X[2] = d_State[tid + 2*N];
		
		P = d_Parameters[tid];
		
		for (int i=0; i<1000; i++)
		{
			// k1
			Lorenz(k1, X, P);
			
			// k2
			//t = T + 0.5*dT;
			
			#pragma unroll 3
			for (int j=0; j<3; j++)
			{
				x[j]  = X[j] + dTp2*k1[j];
				ks[j] = k1[j];
			}
			
			Lorenz(k1, x, P);
			
			// k3
			//t = T + 0.5*dT;
			
			#pragma unroll 3
			for (int j=0; j<3; j++)
			{
				x[j]  = X[j] + dTp2*k1[j];
				ks[j] = ks[j]+2*k1[j];
			}
			
			Lorenz(k1, x, P);
			
			// k4
			//t = T + dT;
			
			#pragma unroll 3
			for (int j=0; j<3; j++)
			{
				x[j] = X[j] + dT*k1[j];
				ks[j] = ks[j]+2*k1[j];
			}
			
			Lorenz(k1, x, P);
			
			
			// Update state
			#pragma unroll 3
			for (int j=0; j<3; j++)
				X[j] = X[j] + dTp6*( ks[j] + k1[j] );
			
			T += dT;
		}
		
		d_State[tid] = X[0];
		d_State[tid + N] = X[1];
		d_State[tid + 2*N] = X[2];
	}
}

template <class DataType>
DataType* AllocateHostMemory(int N)
{
    DataType* HostMemory = new (std::nothrow) DataType [N];
    if (HostMemory == NULL)
    {
        std::cerr << "Failed to allocate Memory on the HOST!\n";
        exit(EXIT_FAILURE);
    }
    return HostMemory;
}

template <class DataType>
DataType* AllocateDeviceMemory(int N)
{
    hipError_t Error = hipSuccess;
	
	DataType* MemoryAddressInDevice = NULL;
	
	Error = hipMalloc((void**)&MemoryAddressInDevice, N * sizeof(DataType));
    
	if (Error != hipSuccess)
    {
        std::cerr << "Failed to allocate Memory on the DEVICE!\n";
        exit(EXIT_FAILURE);
    }
    return MemoryAddressInDevice;
}