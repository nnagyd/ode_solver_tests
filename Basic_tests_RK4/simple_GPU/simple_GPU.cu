
#include <hip/hip_runtime.h>
#include <iostream>
#include <iomanip>
#include <vector>
#include <string>
#include <fstream>

using namespace std;

void Linspace(double*, double, double, int);
void Uniform(double*, double, int);

__global__ void RungeKuttaStepOriginal(double*, double*, int);
__device__ void RightHandSide(double&, double, double);

template <class DataType>
DataType* AllocateHostMemory(int);
template <class DataType>
DataType* AllocateDeviceMemory(int);

int main()
{
// INITIAL SETUP ----------------------------------------------------------------------------------
	
	int NumberOfProblems = 61440; // 92160
	int BlockSize        = 128;
	
	hipSetDevice(1);
	
	double* h_State      = AllocateHostMemory<double>(NumberOfProblems);
	double* h_Parameters = AllocateHostMemory<double>(NumberOfProblems);
	double* d_State      = AllocateDeviceMemory<double>(NumberOfProblems);
	double* d_Parameters = AllocateDeviceMemory<double>(NumberOfProblems);
	
	Linspace(h_Parameters, 0.1, 1.0, NumberOfProblems);
	Uniform(h_State, -0.5, NumberOfProblems);
	
	hipMemcpy(d_State, h_State, sizeof(double)*NumberOfProblems, hipMemcpyHostToDevice);
	hipMemcpy(d_Parameters, h_Parameters, sizeof(double)*NumberOfProblems, hipMemcpyHostToDevice);
	
	
	int GridSize = NumberOfProblems/BlockSize + (NumberOfProblems % BlockSize == 0 ? 0:1);
	
	clock_t SimulationStart;
	clock_t SimulationEnd;
	
	SimulationStart = clock();
	RungeKuttaStepOriginal<<<GridSize, BlockSize>>> (d_State, d_Parameters, NumberOfProblems);
	SimulationEnd = clock();
	
	cout << "Simulation time: " << 1000.0*(SimulationEnd-SimulationStart) / CLOCKS_PER_SEC << "ms" << endl << endl;
	cout << "Simulation time / 1000 RK4 step: " << 1000.0*(SimulationEnd-SimulationStart) / CLOCKS_PER_SEC << "ms" << endl;
	cout << "Ensemble size:                   " << NumberOfProblems << endl << endl;
		
	hipMemcpy(h_State, d_State, sizeof(double)*NumberOfProblems, hipMemcpyDeviceToHost);
	
	//for (int i=0; i<NumberOfProblems; i++)
	//	cout << "P: " << h_Parameters[i] << "   Sates: " << h_State[i] << endl;
}

// AUXILIARY FUNCTION -----------------------------------------------------------------------------

void Linspace(double* x, double B, double E, int N)
{
    double Increment;
	
	x[0]   = B;
	
	if ( N>1 )
	{
		x[N-1] = E;
		Increment = (E-B)/(N-1);
		
		for (int i=1; i<N-1; i++)
		{
			x[i] = B + i*Increment;
		}
	}
}

void Uniform(double* x, double V, int N)
{
	for (int i=0; i<N; i++)
	{
		x[i] = V;
	}
}

__forceinline__ __device__ void RightHandSide(double& F, double X, double P)
{
	F = X*X - P; // 1 FMA
}

__global__ void RungeKuttaStepOriginal(double* d_State, double* d_Parameters, int N)
{
	int tid = threadIdx.x + blockIdx.x*blockDim.x;
	
	if (tid < N)
	{
		double X;
		double P;
		
		double k1;
		double k2;
		double k3;
		double k4;
		double x;
		
		double dT   = 0.01;
		double dTp2 = 0.5*dT;
		double dTp6 = dT * (1.0/6.0);
		
		X = d_State[tid];
		P = d_Parameters[tid];
		
		for (int i=0; i<1000; i++)
		{
			// k1
			RightHandSide(k1, X, P);
			
			x = X + dTp2*k1;
			RightHandSide(k2, x, P);
			
			x = X + dTp2*k2;
			RightHandSide(k3, x, P);
			
			x = X + dT*k3;
			RightHandSide(k4, x, P);
			
			X = X + dTp6*( k1 + 2*k2 + 2*k3 + k4 );
		}
		
		d_State[tid] = X;
	}
}

template <class DataType>
DataType* AllocateHostMemory(int N)
{
    DataType* HostMemory = new (std::nothrow) DataType [N];
    if (HostMemory == NULL)
    {
        std::cerr << "Failed to allocate Memory on the HOST!\n";
        exit(EXIT_FAILURE);
    }
    return HostMemory;
}

template <class DataType>
DataType* AllocateDeviceMemory(int N)
{
    hipError_t Error = hipSuccess;
	
	DataType* MemoryAddressInDevice = NULL;
	
	Error = hipMalloc((void**)&MemoryAddressInDevice, N * sizeof(DataType));
    
	if (Error != hipSuccess)
    {
        std::cerr << "Failed to allocate Memory on the DEVICE!\n";
        exit(EXIT_FAILURE);
    }
    return MemoryAddressInDevice;
}